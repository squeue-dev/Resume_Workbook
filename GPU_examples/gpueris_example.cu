#include "hip/hip_runtime.h"
/* Code example for the evaluation of quantum Chemical Electron Repulsion 
   Integrals solved analytically through Obara and Saika's Recurrence eqs.
   
   Martin Head‐Gordon and John A. Pople, A method for two‐electron Gaussian
   integral and integral derivative evaluation using recurrence relations, 
   J. Chem. Phys. 89, 5777-5786 (1988) https://doi.org/10.1063/1.455553

Author: Alfonso Esqueda García, esqueda.alfonso.94@gmail.com
Year: 2018
*/  




#include <cstdlib>
#include <iostream>
#include <stdio.h>
#include <math.h>
#include <ctime>

//Linear algebra libraries
#include "hipblas.h"
#include "magma_v2.h"
#include "magma_lapack.h"

#define BOYS_MIN 1.0e-13
#define GALLETA_MAX_L_I 8
#define GALLETA_MAX_TAB_GAM 120
#define GALLETA_ABS(a) ((a)>=0.0?(a):-(a))
#define GALLETA_PI M_PI

using namespace std;
using std::cout;
using std::endl;


__device__ double F0(double t){

  if (t < 1.0E-6){
    //ASYMPTOTIC VALUE FOR SMALL ARGUMENTS
    return 1.0 - t / 3.0;
  }
  else{
    //F0 IN TERMS OF THE ERROR FUNCTION
    return 0.5*pow(3.1416/t,1.0/2.0)*erf(pow(t,1.0/2.0));
  }
}

// AEG: Function to contract the primitive integrals from within the GPU for -#
//      any contracted integral. ---------------------------------------------#
// Note: In the middle of creating a function to contract each type of eri ---#
//       I noticed there was a way to create a general function to do that ---#
//       regardless of the type of eri to cotract ----------------------------#
__global__ void primi2contr(int *shellidxs_ll_a, int * shellidxs_ul_a,
                            int *shellidxs_ll_b, int * shellidxs_ul_b,
                            double *primeri, double *contreri,
                            double *coef_a, double *coef_b, int nshells_a,
                            int nshells_b, int nprimis, int auxnco,
                            int ori1, int ori2, int ori3, double *ncsto_a, double *ncsto_b, double *ncsto_aux)
{
  for (int i=blockIdx.x * blockDim.x + threadIdx.x; i<nshells_a; i+=blockDim.x * gridDim.x) {
    for (int j=blockIdx.y * blockDim.y + threadIdx.y; j<nshells_b; j+=blockDim.y * gridDim.y) {
      for (int r=blockIdx.z * blockDim.z + threadIdx.z; r<auxnco; r+=blockDim.z * gridDim.z) {

        int nori = ori1*ori2*ori3;

        int lli = shellidxs_ll_a[i];
        int uli = shellidxs_ul_a[i];

        int llj = shellidxs_ll_b[j];
        int ulj = shellidxs_ul_b[j];

        for (int ctr_aux=0; ctr_aux<ori3; ctr_aux++) {
          for (int ctr_a=0; ctr_a<ori1; ctr_a++) {
            for (int ctr_b=0; ctr_b<ori2; ctr_b++) {
              double sum = 0.0;
              for (int k=lli; k<=uli; k+=ori1) {
                for (int l=llj; l<=ulj; l+=ori2) {
                  sum += primeri[(k+ctr_a)*nprimis*ori2*auxnco*ori3+(l+ctr_b)*auxnco*ori3+r*ori3+ctr_aux] *
                         coef_a[k/ori1] * coef_b[l/ori2];
                }
              }
              contreri[(i*ori1+ctr_a)*nshells_b*ori2*auxnco*ori3+(j*ori2+ctr_b)*auxnco*ori3+(r*ori3+ctr_aux)] = sum*ncsto_a[i*ori1+ctr_a]*ncsto_b[j*ori2+ctr_b]*ncsto_aux[ori3*r+ctr_aux];
            }
          }
        }

      }
    }
  }
}
// ---------------------------------------------------------------------------#
// ---------------------------------------------------------------------------#


// AEG: Function to add the contributions of the contracted eris to the Fock -#
//      matrix. --------------------------------------------------------------#
__global__ void contr2FockM(double *contreri, double *FockM, int nshells,
                            int n_shells_a, int n_shells_b, int n_aux, double *w, double sf, int *ll_ao_shell_a, int *ll_ao_shell_b, int nbas, int ori1, int ori2, int ori3)
{
  double sum;
  double factor;
  int ii, jj;
  //Falta considerar las orientaciones, o quiza si estan, hay que ver como se calcula nshells
  for (int i=blockIdx.x * blockDim.x + threadIdx.x; i<n_shells_a; i+=blockDim.x * gridDim.x) {
    ii = ll_ao_shell_a[i];
    for (int j=blockIdx.y * blockDim.y + threadIdx.y; j<n_shells_b; j+=blockDim.y * gridDim.y) {
      jj = ll_ao_shell_b[j];
        for (int iori1=0; iori1<ori1; iori1++) {
          int iii = ii + iori1;
          for (int jori2=0; jori2<ori2; jori2++) {
            int jjj = jj + jori2;
            sum = 0.0;
            for (int k=0; k<n_aux; k++) {
              for (int kori3=0; kori3<ori3; kori3++) {
                int nori = ori1*ori2*ori3;
                factor = w[k*ori3+kori3] * sf;
                sum += factor * contreri[(i*ori1+iori1)*n_shells_b*ori2*n_aux*ori3+(j*ori2+jori2)*n_aux*ori3+(k*ori3+kori3)];
            }
          }
          FockM[iii*nbas+jjj] += sum;
        }
      }
    }
  }
}
// ---------------------------------------------------------------------------#
// ---------------------------------------------------------------------------#



// AEG: Computes the integral (s|s) needed for the normalization of auxiliary-#
// basis.---------------------------------------------------------------------#
__device__ void ss(double *a,double *b, double exp_a, double exp_b,
                   double coef_a, double coef_b, int la, int lb,
                   double &eri) {
  //From Szabo Apendix A p 410 y 416

  double Rab2, coef, f0;

  // Compute ERI
  coef = 2*pow(3.1416,5.0/2.0)/(exp_a*exp_b*pow(exp_a+exp_b,1.0/2.0));

  Rab2 = pow(a[0]-b[0],2) + pow(a[1]-b[1],2) + pow(a[2]-b[2],2);

  f0 = F0(exp_a*exp_b/(exp_a+exp_b)*Rab2);

  eri = coef*f0;
}
// ---------------------------------------------------------------------------#
// ---------------------------------------------------------------------------#



// AEG: Factorial function needed for the Evaluation of the Boys function ----#
//      or "Gamma Function" routien ----------------------------------- ------#
__device__ int factorial(int n) {
    if(n > 1)
        return n * factorial(n - 1);
    else
        return 1;
}
// ---------------------------------------------------------------------------#
// ---------------------------------------------------------------------------#



// AEG: Initialization of the tabulated values needed for the evaluation of --#
//      the Boys function ----------------------------------------------------#
//
// *This function can be device, global or called by host depending of needs
//  with just a little modification
//
// *ftab needs to be of dimensions:
//  ftab[2*GALLETA_MAX_L_I+6+1][GALLETA_MAX_TAB_GAM+1] -----------------------#
__device__ void init_boysfunc(double ftab[][GALLETA_MAX_TAB_GAM+1]) {
  int i, j, k, l;
  const int nitermax = 30; // needs to be a constant for the nvcc compiler to
                           // accept it as the dimension for r[]
  int nmax = 2 * GALLETA_MAX_L_I + 6;
  double eps = 1.0e-15;
  double bessel, expterm, prefak, preterm, produkt, serie, sumterm, term, ttab;
  double r[nitermax+11];

  for (i=0; i<=nmax; i++)
    ftab[i][0] = 1.0/(2*i+1);
  for (i=1; i<=GALLETA_MAX_TAB_GAM; i++) {
    ttab = double(i)/10.0;
    r[nitermax+10] = 0.0;
    for (j=1; j<=nitermax+9; j++)
      r[nitermax+10-j] = -ttab/(4*(nitermax+10-j) + 2.0 - ttab *  r[nitermax+11-j]);
    bessel = (2 * sinh(ttab / 2)) / ttab;
    prefak = exp(-ttab / 2) * bessel;
    term = 1.0;
    serie = prefak * (1.0 / (2.0 * nmax + 1.0));
    for (k=1; k<=nitermax; k++) {
      preterm = (2.0 * k + 1.0) / (2.0 * nmax + 1.0);
      term = term * (2.0 * nmax - 2.0 * k + 1.0) / (2.0 * nmax + 2.0 * k + 1.0);
      produkt = 1.0;
      for (l=1; l<=k; l++)
        produkt = produkt * r[l];
      sumterm = prefak * preterm * term * produkt;
      if (GALLETA_ABS(sumterm)<=eps) goto TABOK;
      else serie = serie + sumterm;
    }
TABOK:
    ftab[nmax][i] = serie;
    expterm = exp(-ttab);
    for (j=1; j<=nmax; j++)
      ftab[nmax-j][i] = 1.0 / (2 * (nmax - j) + 1) * (2 * ttab * ftab[nmax+1-j][i] + expterm);
  }
}
// ---------------------------------------------------------------------------#
// ---------------------------------------------------------------------------#



// AEG: Calculation of the Boys function F(t) for bielectronic integrals over-#
// Gaussian functions. -------------------------------------------------------#
//
// Taken from Green.128.
// Originally taken from:
// L. E. McMurchie and E. R. Davidson,
// J. Comp. Phys. 26, 218 (1978).
// ---------------------------------------------------------------------------#
// ---------------------------------------------------------------------------#
__device__ void boysfunc(int m, double& t, double* fis,// vector with F(i)T
                          double ftab[][GALLETA_MAX_TAB_GAM+1]) {
  int i, k, ttab;
  double a, b, c, d, expterm;

  if (t < 0.0) t = BOYS_MIN;
  if (t <= BOYS_MIN) {
    fis[m] = 1.0/ (2.0 * m + 1.0);
    for (i=1; i<=m; i++)
      fis[m-i] = 1.0/(2.0 * (m-i) + 1.0);
    return;
  }
  else if (t <= 12.0) {
    ttab = int(10 * t + 0.5);
    fis[m] = ftab[m][ttab];
    for (k=1; k<=6; k++)
      fis[m] += ftab[m+k][ttab] * (pow(double(ttab) / 10.0 - t, k)) / factorial(k);
    if (m > 0) expterm = exp(-t);
    for (i=1; i<=m; i++)
      fis[m-i] = 1.0 / (2 * (m - i) + 1) * (2 * t * fis[m+1-i] + expterm);
    return;
  }
  else if (t <= 15.0) {
    a = 0.4999489092;
    b = 0.2473631686;
    c = 0.3211809090;
    d = 0.3811559346;
    fis[0] = 0.5 * sqrt(GALLETA_PI / t) - (exp(-t) / t) * (a - b / t + c / (t * t) - d / (t * t * t));
  }
  else if (t <= 18.0) {
    a = 0.4998436875;
    b = 0.2424943800;
    c = 0.2464284500;
    fis[0] = 0.5 * sqrt(GALLETA_PI / t) - (exp(-t) / t) * (a - b / t + c / (t * t));
  }
  else if (t <= 24.0) {
    a = 0.4990931620;
    b = 0.2152832000;
    fis[0] = 0.5 * sqrt(GALLETA_PI / t) - (exp(-t) / t) * (a - b / t);
  }
  else if (t <= 30.0) {
    a = 0.49000000;
    fis[0] = 0.5 * sqrt(GALLETA_PI / t) - (exp(-t) / t) * a;
  }
  else {
    fis[0] = 0.5 * sqrt(GALLETA_PI / t);
  }
  if (t > (2.0 * m + 36)) {
    for (i=1; i<=m; i++)
      fis[i] = (2 * i - 1) / (2 * t) * fis[i-1];
  }
  else {
    expterm = exp(-t);
    for (i=1; i<=m; i++)
      fis[i] = 1 / (2 * t) * ((2 * i -1) * fis[i-1] - expterm);
  }
}
// ---------------------------------------------------------------------------#
// ---------------------------------------------------------------------------#



// AEG: Function to compute a three center integral of the type (ss|p) -------#
//      using the formula obtained from aplying the OS RR method. ------------#
__global__ void ssp(double *a, double *b, double *caux, double *exp_a,
                    double *exp_b, double *exp_caux, double *coef_a,
                    double *coef_b, double *coef_caux, int la, int lb,
                    int lcaux, int nprimis, int auxnco, double *primeri,
                    int *atomidx, int *atomidxaux) {

  double Rab2, Rpcaux2, Kab, p, coef, t, eri;
  double Rp[3], Rw[3], fis[2], eri_v[3];
  double caux4norm[3];
  int atidxa, atidxb, atidxaux;
  double normceri = 0.0;

  __shared__ double ftab[2*GALLETA_MAX_L_I+6+1][GALLETA_MAX_TAB_GAM+1];

  if (threadIdx.x == 0 && threadIdx.y == 0 && threadIdx.z == 0) {
    // Initialize ftab
    init_boysfunc(ftab);
  }
  __syncthreads();

  // AEG loop for offsetting
  for (int x=blockIdx.x * blockDim.x + threadIdx.x; x<nprimis; x+=blockDim.x * gridDim.x) {
    for (int y=blockIdx.y * blockDim.y + threadIdx.y; y<nprimis; y+=blockDim.y * gridDim.y) {
      for (int z=blockIdx.z * blockDim.z + threadIdx.z; z<auxnco; z+=blockDim.z * gridDim.z) {
        atidxa = atomidx[x];
        atidxb = atomidx[y];
        atidxaux = atomidxaux[z];

        fis[0] = 0.0;
        fis[1] = 0.0;

        // Calculation of [ss||s]^(1)

          // Bra contraction using gaussian product theorem
        Rab2 = pow(a[atidxa*3]-b[atidxb*3],2) + pow(a[atidxa*3+1]-b[atidxb*3+1],2) + pow(a[atidxa*3+2]-b[atidxb*3+2],2);
        Kab = exp(-((exp_a[x]*exp_b[y]/(exp_a[x]+exp_b[y]))*(Rab2)));
        p = exp_a[x] + exp_b[y];
        Rp[0] = (exp_a[x]*a[atidxa*3]+exp_b[y]*b[atidxb*3])/(exp_a[x]+exp_b[y]);
        Rp[1] = (exp_a[x]*a[atidxa*3+1]+exp_b[y]*b[atidxb*3+1])/(exp_a[x]+exp_b[y]);
        Rp[2] = (exp_a[x]*a[atidxa*3+2]+exp_b[y]*b[atidxb*3+2])/(exp_a[x]+exp_b[y]);

          // Compute ERI
        coef = 2*pow(GALLETA_PI,5.0/2.0)/(p*exp_caux[z]*pow(p+exp_caux[z],1.0/2.0));
        Rpcaux2 = pow(Rp[0]-caux[atidxaux*3],2) + pow(Rp[1]-caux[atidxaux*3+1],2) + pow(Rp[2]-caux[atidxaux*3+2],2);
          // Boys function with m=1
        t = ((p * exp_caux[z]) / (p + exp_caux[z])) * Rpcaux2;
        boysfunc(1, t, fis, ftab);

        eri = coef * Kab * fis[1];

        caux4norm[0] = caux[atidxaux*3];
        caux4norm[1] = caux[atidxaux*3+1];
        caux4norm[2] = caux[atidxaux*3+2];

        // Multiply [ss||s]^(1) by (Wi-Ci)
        Rw[0] = (p * Rp[0] + exp_caux[z] * caux[atidxaux*3]) / (p + exp_caux[z]);
        Rw[1] = (p * Rp[1] + exp_caux[z] * caux[atidxaux*3+1]) / (p + exp_caux[z]);
        Rw[2] = (p * Rp[2] + exp_caux[z] * caux[atidxaux*3+2]) / (p + exp_caux[z]);

        eri_v[0] = (Rw[0] - caux[atidxaux*3]) * eri;
        eri_v[1] = (Rw[1] - caux[atidxaux*3+1]) * eri;
        eri_v[2] = (Rw[2] - caux[atidxaux*3+2]) * eri;

        primeri[x*nprimis*3*auxnco+y*3*auxnco+3*z+0] = eri_v[0];
        primeri[x*nprimis*3*auxnco+y*3*auxnco+3*z+1] = eri_v[1];
        primeri[x*nprimis*3*auxnco+y*3*auxnco+3*z+2] = eri_v[2];

      }
    }
  }
}
// ---------------------------------------------------------------------------#
// ---------------------------------------------------------------------------#
